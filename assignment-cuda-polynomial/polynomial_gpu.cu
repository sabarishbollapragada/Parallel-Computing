
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>


__global__ void polynomial_expansion (float* poly, int degree,
			   int n, float* array) {
  //TODO: Write code to use the GPU here!
  //code should write the output back to array
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < n){
    float out = 0.;
    float xtothepowerof = 1.;
    for (int i=0; i<=degree; ++i) {
      out += xtothepowerof*poly[i];
      xtothepowerof *= array[index];
    }
    array[index] = out;

  }

}


int main (int argc, char* argv[]) {
  //TODO: add usage
  
  if (argc < 3) {
     std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
     return -1;
  }

  int n = atoi(argv[1]); //TODO: atoi is an unsafe function
  int degree = atoi(argv[2]);
  int nbiter = 1;

  float* array = new float[n];
  float* poly = new float[degree+1];
  float* gArray;
  float* gPoly;
  for (int i=0; i<n; ++i)
    array[i] = 1.;

  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.;


  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();

  hipMallocManaged(&gArray, n*sizeof(float));
  hipMallocManaged(&gPoly, (degree+1)*sizeof(float));

  hipMemcpy(gArray,array,n*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(gPoly,poly,(degree+1)*sizeof(float),hipMemcpyHostToDevice);

  int blocks=n/256;
  if(n%256>0){
    blocks = n/256+1;
  }

  
  
  for (int iter = 0; iter<nbiter; ++iter)
    polynomial_expansion<<<blocks,256>>> (gPoly, degree, n, gArray);

  

  hipMemcpy(array,gArray,n*sizeof(float),hipMemcpyDeviceToHost);
  hipFree(gArray);
  hipFree(gPoly);

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin)/nbiter;

  {
    bool correct = true;
    int ind;
    for (int i=0; i< n; ++i) {
      if (fabs(array[i]-(degree+1))>0.01) {
        correct = false;
	ind = i;
      }
    }
    if (!correct)
      std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
  }
  

  std::cerr<<array[0]<<std::endl;
  std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

  delete[] array;
  delete[] poly;

  return 0;
}
